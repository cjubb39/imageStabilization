#include "hip/hip_runtime.h"
#include "error_handling.h"
//#include "im1.h"

#define rowmajIndex(col, row, width, height) ( ((int) row + height/2)*width + ((int) col + width/2))


/*	
 *	transform_info: cos(angle), sin(angle), trans_x, trans_y
 *	
 */
__global__ void image_transform(float *source, float *destination,
		int width, int height, int xtrans, int ytrans, int dwidth, int dheight,
		float *transform_info){

	/* want origin at center */
	const int x = blockIdx.x * blockDim.x + threadIdx.x - dwidth / 2;
	const int y = blockIdx.y * blockDim.y + threadIdx.y - dheight / 2;
	const int index = rowmajIndex(x, y, dwidth, dheight);

	if (x >= dwidth/2 || y >= dheight/2){
		return;
	}

	/* do translation */
	float fetch_x = x + xtrans + transform_info[2];
	float fetch_y = y + ytrans + transform_info[3];

	/* do rotation */
	float cos_val = transform_info[0], sin_val = transform_info[1];
	
	int tmp = fetch_x;
	fetch_x = tmp*cos_val - fetch_y*sin_val;
	fetch_y = tmp*sin_val + fetch_y*cos_val;

/*printf("Coord: %d, %d: %d; Dim: %d %d" ";; fetch: %d, %d: %d\n", 
	x, y, 3*rowmajIndex(x,y,width, height), width, height, 
	(int) fetch_x, (int) fetch_y, (int) (3*rowmajIndex((int) fetch_x, (int) fetch_y,width, height)));
*/
	if (fetch_x >= width/2 || fetch_x < -width/2 ||
			fetch_y >= height/2 || fetch_y < -height/2){
		destination[3*index] = 0.25;
		destination[3*index + 1] = 0.25;
		destination[3*index + 2] = 0.25;
	} else {
		destination[3*index] =
			source[(int) (3 * rowmajIndex(fetch_x, fetch_y, width, height))];
		destination[3*index + 1] =
			source[(int) (3 * rowmajIndex(fetch_x, fetch_y, width, height) + 1)];
		destination[3*index + 2] =
			source[(int) (3 * rowmajIndex(fetch_x, fetch_y, width, height) + 2)];
	}
}

__host__ void getMaxThreadsPerBlock(int *info){
    int dev_count, highTPB = 0, highdev = 0;
    GPU_CHECKERROR(hipGetDeviceCount(&dev_count));
    for (int dev = 0; dev < dev_count; dev++){
        hipDeviceProp_t deviceProp;
        GPU_CHECKERROR(hipGetDeviceProperties(&deviceProp, dev));

        if (deviceProp.maxThreadsPerBlock > highTPB){
            highTPB = deviceProp.maxThreadsPerBlock;
            highdev = dev;
        }
    }

    info[0] = highdev;
    info[1] = highTPB;
}


/*	
 *	transform should be in form: (single pointer to array in row major form)
 *		cos(theta)	-sin(theta)	t_x
 *		sin(theta)	cos(theta)	t_y
 *		0			0			1
 */
__host__ void apply_transform(float *input, float *output, double *transform,
		const int width, const int height, int xtrans, int ytrans,
		int dwidth, int dheight){

	 // start the timers
  hipEvent_t     start, stop;
  float elapsedTime;
  GPU_CHECKERROR( hipEventCreate( &start ) );
  GPU_CHECKERROR( hipEventCreate( &stop ) );
  GPU_CHECKERROR( hipEventRecord( start, 0 ));


	/* convert transform to whats used below */
	float tmp_transform[4];
	tmp_transform[0] = transform[0];
	tmp_transform[1] = -transform[1];
	tmp_transform[2] = transform[2];
	tmp_transform[3] = transform[5];

#ifdef DEBUG
printf("transform prop: %f %f %f %f\n", tmp_transform[0], tmp_transform[1], tmp_transform[2], tmp_transform[3]);
#endif

	float *d_source, *d_destination, *d_transform_info;
	GPU_CHECKERROR(hipMalloc(&d_source, sizeof(float) * height * width * 3));
	GPU_CHECKERROR(hipMalloc(&d_destination, sizeof(float) * dheight * dwidth * 3));
	GPU_CHECKERROR(hipMalloc(&d_transform_info, sizeof(float) * 4));

	GPU_CHECKERROR(hipMemcpy(d_source, input, 
		sizeof(float) * height * width * 3, hipMemcpyHostToDevice));
	GPU_CHECKERROR(hipMemcpy(d_destination, output, sizeof(float) * dheight * dwidth * 3,
			hipMemcpyHostToDevice));
	GPU_CHECKERROR(hipMemcpy(d_transform_info, tmp_transform,
		sizeof(float) * 4, hipMemcpyHostToDevice));

	/* run kernel */
	int device_info[2];
	getMaxThreadsPerBlock(device_info);
	GPU_CHECKERROR(hipSetDevice(device_info[0]));

	int threads_per_block = device_info[1];

	dim3 block_size, grid_size;
	block_size.x = 32; // warp size
	block_size.y = (unsigned int) (threads_per_block / block_size.x);

	grid_size.x = (unsigned int) ((dwidth + block_size.x - 1) / block_size.x);
    grid_size.y = (unsigned int) ((dheight + block_size.y - 1) / block_size.y);

    image_transform<<<grid_size, block_size>>>
    		(d_source, d_destination, width, height, xtrans, ytrans, dwidth,
    				dheight, d_transform_info);
    GPU_CHECKERROR(hipDeviceSynchronize());

    GPU_CHECKERROR(hipMemcpy(output, d_destination,
    		sizeof(float) * dheight * dwidth * 3, hipMemcpyDeviceToHost));
    GPU_CHECKERROR(hipFree(d_source));
    GPU_CHECKERROR(hipFree(d_destination));
    GPU_CHECKERROR(hipFree(d_transform_info));

    GPU_CHECKERROR( hipEventRecord( stop, 0 ));
    GPU_CHECKERROR( hipEventSynchronize( stop ) );
    GPU_CHECKERROR( hipEventElapsedTime( &elapsedTime,
                                      start, stop ) );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );
}

/* Given the set of transformation matrices, find the final destination
 * translation wrt to the original, and size
 */
__host__ void find_dest_multi(double *transforms, int num_transforms, int width,
		int height, int *xtrans, int *ytrans, int *dwidth, int *dheight)
{
	//Start with assuming no transformation
	int xmin = 0;
	int ymin = 0;
	int xmax = width;
	int ymax = height;

	for (int i = 0; i < num_transforms; i++)
	{
		int index = i*9;
		//Find the farthest transform that's above and to the left
		if (transforms[index + 2] < *xtrans)
			*xtrans = transforms[index + 2];
		if (transforms[index + 5] < *ytrans)
			*ytrans = transforms[index + 5];

		//Translate the four corners to find the largest dimensions
		int x1 = -width/2 * transforms[index]
		         - height/2 * transforms[index + 1] + transforms[index + 2] + width/2;
		int y1 = -width/2 * transforms[index + 3]
		         - height/2 * transforms[index + 4] + transforms[index + 5] + height/2;
		int x2 = -width/2 * transforms[index]
		         + height/2 * transforms[index + 1] + transforms[index + 2] + width/2;
		int y2 = -width/2 * transforms[index + 3]
		         + height/2 * transforms[index + 4] + transforms[index + 5] + height/2;
		int x3 = width/2 * transforms[index]
		         - height/2 * transforms[index + 1] + transforms[index + 2] + width/2;
		int y3 = width/2 * transforms[index + 3]
		         - height/2 * transforms[index + 4] + transforms[index + 5] + height/2;
		int x4 = width/2 * transforms[index]
		         + height/2 * transforms[index + 1] + transforms[index + 2] + width/2;
		int y4 = width/2 * transforms[index + 3]
		         + height/2 * transforms[index + 4] + transforms[index + 5] + height/2;

		xmin = min(min(min(min(x1, x2), x3), x4), xmin);
		ymin = min(min(min(min(y1, y2), y3), y4), ymin);
		xmax = max(max(max(max(x1, x2), x3), x4), xmax);
		ymax = max(max(max(max(y1, y2), y3), y4), ymax);
	}

	*xtrans = -(*xtrans)/2;
	*ytrans = -(*ytrans)/2;

	*dwidth = xmax - xmin;
	*dheight = ymax - ymin;
}
