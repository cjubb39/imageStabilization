#include "hip/hip_runtime.h"
#include "error_handling.h"
#include "match.h"
#include "im1.h"

#define rowmajIndex(col, row, width, height) ( ((int) row + height/2)*width + ((int) col + width/2))

/*	
 *	transform_info: cos(angle), sin(angle), trans_x, trans_y
 *	
 */
__global__ void image_transform(float *source, float *destination, 
		int width, int height, float *transform_info){

	/* want origin at center */
	const int x = blockIdx.x * blockDim.x + threadIdx.x - width / 2;
	const int y = blockIdx.y * blockDim.y + threadIdx.y - height / 2;
	const int index = rowmajIndex(x, y, width, height);

	if (x >= (width / 2) || y >= (height / 2)){
		return;
	}

	/* do translation */
	float fetch_x = x - transform_info[2];
	float fetch_y = y + transform_info[3];

	/* do rotation */
	float cos_val = transform_info[0], sin_val = transform_info[1];
	
	int tmp = fetch_x;
	fetch_x = tmp*cos_val - fetch_y*sin_val;
	fetch_y = tmp*sin_val + fetch_y*cos_val;

/*printf("Coord: %d, %d: %d; Dim: %d %d" ";; fetch: %d, %d: %d\n", 
	x, y, 3*rowmajIndex(x,y,width, height), width, height, 
	(int) fetch_x, (int) fetch_y, (int) (3*rowmajIndex((int) fetch_x, (int) fetch_y,width, height)));
*/
	if (fetch_x >= width/2 || fetch_x <= -width/2 || 
			fetch_y >= height/2 || fetch_y <= -height/2){
		destination[3*index] = 0;
		destination[3*index + 1] = 1;
		destination[3*index + 2] = 0;
	} else {
		destination[3 * rowmajIndex(x,y,width, height)] =
			source[(int) (3 * rowmajIndex(fetch_x, fetch_y, width, height))];
		destination[3 * rowmajIndex(x,y,width, height) + 1] =
			source[(int) (3 * rowmajIndex(fetch_x, fetch_y, width, height) + 1)];
		destination[3 * rowmajIndex(x,y,width, height) + 2] =
			source[(int) (3 * rowmajIndex(fetch_x, fetch_y, width, height) + 2)];
	}
}

__host__ void getMaxThreadsPerBlock(int *info){
    int dev_count, highTPB = 0, highdev = 0;
    GPU_CHECKERROR(hipGetDeviceCount(&dev_count));
    for (int dev = 0; dev < dev_count; dev++){
        hipDeviceProp_t deviceProp;
        GPU_CHECKERROR(hipGetDeviceProperties(&deviceProp, dev));

        if (deviceProp.maxThreadsPerBlock > highTPB){
            highTPB = deviceProp.maxThreadsPerBlock;
            highdev = dev;
        }
    }

    info[0] = highdev;
    info[1] = highTPB;
}


/*	
 *	transform should be in form: (single pointer to array in row major form)
 *		cos(theta)	-sin(theta)	t_x
 *		sin(theta)	cos(theta)	t_y
 *		0						0						1
 */
__host__ void apply_transform(float *input, float *output, float *transform, 
		const int width, const int height){

	 // start the timers
  hipEvent_t     start, stop;
  float elapsedTime;
  GPU_CHECKERROR( hipEventCreate( &start ) );
  GPU_CHECKERROR( hipEventCreate( &stop ) );
  GPU_CHECKERROR( hipEventRecord( start, 0 ));


	/* convert transform to whats used below */
	float tmp_transform[4];
	tmp_transform[0] = transform[0];
	tmp_transform[1] = -transform[1];
	tmp_transform[2] = transform[2];
	tmp_transform[3] = transform[5];

#ifdef DEBUG
printf("transform prop: %f %f %f %f\n", tmp_transform[0], tmp_transform[1], tmp_transform[2], tmp_transform[3]);
#endif

	float *d_source, *d_destination, *d_transform_info;
	GPU_CHECKERROR(hipMalloc(&d_source, sizeof(float) * height * width * 3));
	GPU_CHECKERROR(hipMalloc(&d_destination, sizeof(float) * height * width * 3));
	GPU_CHECKERROR(hipMalloc(&d_transform_info, sizeof(float) * 4));

	GPU_CHECKERROR(hipMemcpy(d_source, input, 
		sizeof(float) * height * width * 3, hipMemcpyHostToDevice));
	GPU_CHECKERROR(hipMemset(d_destination, 0x00, sizeof(float) * height * width * 3));
	GPU_CHECKERROR(hipMemcpy(d_transform_info, tmp_transform,
		sizeof(float) * 4, hipMemcpyHostToDevice));

	/* run kernel */
	int device_info[2];
	getMaxThreadsPerBlock(device_info);
	GPU_CHECKERROR(hipSetDevice(device_info[0]));

	int threads_per_block = device_info[1];

	dim3 block_size, grid_size;
	block_size.x = 32; // warp size
	block_size.y = (unsigned int) (threads_per_block / block_size.x);

	grid_size.x = (unsigned int) ((width + block_size.x - 1) / block_size.x);
  grid_size.y = (unsigned int) ((height + block_size.y - 1) / block_size.y);

  image_transform<<<grid_size, block_size>>>
  	(d_source, d_destination, width, height, d_transform_info);
  GPU_CHECKERROR(hipDeviceSynchronize());

  GPU_CHECKERROR(hipMemcpy(output, d_destination,
  	sizeof(float) * height * width * 3, hipMemcpyDeviceToHost));
  GPU_CHECKERROR(hipFree(d_source));
  GPU_CHECKERROR(hipFree(d_destination));
  GPU_CHECKERROR(hipFree(d_transform_info));

  GPU_CHECKERROR( hipEventRecord( stop, 0 ));
  GPU_CHECKERROR( hipEventSynchronize( stop ) );
  GPU_CHECKERROR( hipEventElapsedTime( &elapsedTime,
                                      start, stop ) );
  printf( "Time taken:  %3.1f ms\n", elapsedTime );
}

__host__ int main(int argc, char **argv){
	printf("reading openEXR file %s\n", argv[1]);
	
	int width, height;

	float *input;
	readOpenEXRFile(argv[1], &input, width, height);

	float *output = (float *) malloc(sizeof(float) * width * height * 3);

	float *transform = (float *) malloc(sizeof(float) * 9);
	transform[0] = 1;//0.707;
	transform[1] = 0;//-0.707;
	transform[2] = 100;
	transform[3] = 0;//0.707;
	transform[4] = 1;//0.707;
	transform[5] = -100;
	transform[6] = 0;
	transform[7] = 0;
	transform[8] = 1;

	apply_transform(input, output, transform, width, height);

	printf("writing output image trans_out.exr\n");
	writeOpenEXRFile("trans_out.exr", output, width, height);
	
	free(transform);
	free(output);
	free(input);

}
